//#include "scale.h"

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <math.h> /* fabs */
#include <string.h>
#include <stdlib.h>
#include <sstream>
#include <unordered_map>

using namespace std;
#define THREADS_PER_BLOCK 256

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}


void printArray(int *arr, int nov){
    for (int j=0;j<nov;j++)
    {
        cout << j << " " << arr[j] << endl;
    }
}
// __device__ bool check(int marked[], int round, int val){
//   for(int i = 0; i < round; i++){
//     if(marked[i] == val){return false;}
//   }
//   return true;
// }
//
// __device__ void DFS_sparse(int xadj[], int adj[], int marked[], int n,
//          int vert, int start, int &count, int round) //vert: bulundugu konum //start: baslangıc noktası
// {
//     marked[round] = vert;
//
//     int start_index = xadj[vert];
//     int path_length = xadj[vert+1];
//
//     if (n == 0){
//       marked[round] = -1;
//         for(int i = start_index; i < path_length; i++){
//             if(adj[i] == start){
//                 count++;
//                 break;
//             }
//         }
//         return;
//     }
//
//     for(int i=start_index; i < path_length; i++){
//         if(check(marked, round,adj[i])){
//             DFS_sparse(xadj, adj, marked, n-1, adj[i], start, count, round + 1);
//         }
//     }
//     marked[round] = -1;
// }


__global__ void kernel3(int* adj, int* xadj, int* output, int nov){
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if(index < nov){
      //int *marked = new int[n];
      //memset(marked, -1, n * sizeof(int)); // bu belki silinebilir
      int localcount = 0;
      // int round = 0;

      // 0-->
      int s0 = xadj[index];
      int e0 = xadj[index+1];

      for(int i=s0; i < e0; i++){
        // 0 --> 1

        int neighbour_1 = adj[i];
        int s1  = xadj[neighbour_1];
        int e1  = xadj[neighbour_1+1];

        for(int j=s1;j < e1; j++){
          // 0 --> 1 --> 2

          int neighbour_2 =  adj[j];
          if (neighbour_2 == index) continue;
          int s2  = xadj[neighbour_2];
          int e2  = xadj[neighbour_2+1];

          for(int k=s2; k < e2; k++){

            // 0 --> 1 --> 2 --> 3
            int neighbour_3 =  adj[k];
            if (neighbour_3 == index){
              localcount+=1;
              break;
            }
          }
        }
      }
      output[index] = localcount;
    }
}

__global__ void kernel4(int* adj, int* xadj, int* output, int nov){
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if(index < nov){
      //int *marked = new int[n];
      //memset(marked, -1, n * sizeof(int)); // bu belki silinebilir
      int localcount = 0;
      // int round = 0;

      // 0-->
      int s0 = xadj[index];
      int e0 = xadj[index+1];

      for(int i=s0; i < e0; i++){
        // 0 --> 1

        int neighbour_1 = adj[i];
        int s1  = xadj[neighbour_1];
        int e1  = xadj[neighbour_1+1];

        for(int j=s1;j < e1; j++){
          // 0 --> 1 --> 2

          int neighbour_2 =  adj[j];
          //eliminate 0 == 2
          if (neighbour_2 == index) continue;
          int s2  = xadj[neighbour_2];
          int e2  = xadj[neighbour_2+1];

          for(int k=s2; k < e2; k++){

            // 0 --> 1 --> 2 --> 3

            int neighbour_3 =  adj[k];
            //eliminate 3 == 0
            if (neighbour_3 == index) continue;
            // eliminate 3 ==1
            if (neighbour_3 == neighbour_1) continue;
            int s3  = xadj[neighbour_3];
            int e3  = xadj[neighbour_3+1];


            for(int n=s3; n < e3; n++){
              //0 -->1 -->2 -->3 -->4


              int neighbour_4 =  adj[n];
              if (neighbour_4 == index){
                localcount+=1;
                break;
              }
            }
          }
        }
      }
      output[index] = localcount;
    }
}
__global__ void kernel5(int* adj, int* xadj, int* output, int nov){
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if(index < nov){
      //int *marked = new int[n];
      //memset(marked, -1, n * sizeof(int)); // bu belki silinebilir
      int localcount = 0;
      // int round = 0;

      // 0-->
      int s0 = xadj[index];
      int e0 = xadj[index+1];

      for(int i=s0; i < e0; i++){
        // 0 --> 1

        int neighbour_1 = adj[i];
        int s1  = xadj[neighbour_1];
        int e1  = xadj[neighbour_1+1];

        for(int j=s1;j < e1; j++){
          // 0 --> 1 --> 2

          int neighbour_2 =  adj[j];
          //eliminate 0 == 2
          if (neighbour_2 == index) continue;
          int s2  = xadj[neighbour_2];
          int e2  = xadj[neighbour_2+1];

          for(int k=s2; k < e2; k++){

            // 0 --> 1 --> 2 --> 3

            int neighbour_3 =  adj[k];
            //eliminate 3 == 0
            if (neighbour_3 == index) continue;
            // eliminate 3 ==1
            if (neighbour_3 == neighbour_1) continue;
            int s3  = xadj[neighbour_3];
            int e3  = xadj[neighbour_3+1];


            for(int n=s3; n < e3; n++){
              //0 -->1 -->2 -->3 -->4


              int neighbour_4 =  adj[n];
              //eliminate 4 == 0
              if (neighbour_4 == index) continue;
              // eliminate 4 ==1
              if (neighbour_4 == neighbour_1) continue;
              // eliminate 4 ==2
              if (neighbour_4 == neighbour_2) continue;

              int s4  = xadj[neighbour_4];
              int e4  = xadj[neighbour_4+1];


              for(int o=s4; o < e4; o++){

                //0 -->1 -->2 -->3 -->4--> 5

                int neighbour_5 =  adj[o];
                if (neighbour_5 == index){
                  localcount+=1;
                  break;
                }
              }
            }
          }
        }
      }
      output[index] = localcount;
    }
}




// __global__ void combination(int* adj, int* xadj, int* output, int n, int nov){
//
//
//   int v =  blockDim.x * blockIdx.x +  threadIdx.x;
//   int i =  blockDim.y * blockIdx.y +  threadIdx.y;
//   int j =  blockDim.z * blockIdx.z +  threadIdx.z;
//
//  // accessVertex
//   // if (v==0 && i == 0 && j ==0) {printf("running");}
//   int start = adj[v];
//   int end   = adj[v+1];
//
//   int neighbourCount = end-start;
//
//
//   if (i>=neighbourCount) return;
//   int first_neighbour = xadj[adj[start + i]];
//
//   int first_neighbour_start = adj[first_neighbour];
//   int first_neighbour_end = adj[first_neighbour+1];
//   int first_neighbour_neighbourCount = first_neighbour_end-first_neighbour_start;
//
//   if(j>=first_neighbour_neighbourCount) return;
//
//   int second_neighbour = xadj[adj[first_neighbour_start+j]];
//
//
//   // check all of them unique
//   if (v == second_neighbour) return;
//
//   // check first -> second -> third has connection
//
//
//   for(int k = start; k < end; k ++){
//      if (xadj[adj[start + k]] == second_neighbour){
//        // AtomicAdd(&output[v],1);
//        break;
//      }
//   }
// }



void wrapper(int *xadj, int *adj, int n,  int nov, int nnz){

  // int X = nov;
  // int Y = maxSize;
  // int Z = maxSize;
  //
  // dim3 threadsPerBlock(8, 8, 8);
  // dim3 numBlocks(X/threadsPerBlock.x,  /* for instance 512/8 = 64*/
  //                Y /threadsPerBlock.y,
  //                Z/threadsPerBlock.z);
  //





  hipSetDevice(0);
  int *adj_d;
  int *xadj_d;
  int *output_d;
  int *output_h = new int[nov];
  int numBlock = (nov + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  hipEvent_t start, stop;
  float elapsedTime;

  gpuErrchk(hipMalloc((void**)&adj_d, (nnz) * sizeof(int)));
  gpuErrchk(hipMalloc((void**)&xadj_d, (nov + 1) * sizeof(int)));

  gpuErrchk(hipMalloc((void**)&output_d, (nov) * sizeof(int)));

  //gpuErrchk(cudaMallocHost((void **)&output_h, (nov) * sizeof(int)));

  gpuErrchk(hipMemcpy(adj_d, adj, (nnz) * sizeof(int), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(xadj_d, xadj, (nov + 1) * sizeof(int), hipMemcpyHostToDevice));

  hipEventCreate(&start);
  hipEventRecord(start, 0);

  kernel5<<<numBlock, THREADS_PER_BLOCK>>>(adj_d, xadj_d, output_d, nov);
  //combination<<<numBlocks, threadsPerBlock>>>(adj_d, xadj_d, output_d, n, nov);

  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipMemcpy(output_h, output_d, (nov) * sizeof(int), hipMemcpyDeviceToHost));

  hipEventCreate(&stop);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&elapsedTime, start, stop);
  // printf("GPU scale took: %f s\n", elapsedTime/1000);
  printArray(output_h,nov);
  hipFree(adj_d);
  hipFree(xadj_d);
}


void  read_mtxbin(string fname, int k){
  //cout << "fname: " << fname << endl;
  ifstream infile(fname);
  int a, b;
  int nnv = 0;
  unordered_map<int, vector<int> > hashmap;

  int  maxElement  = -1;

  while (infile >> a >> b)
  {
      nnv+=2;
      hashmap[a].push_back(b);
      hashmap[b].push_back(a);

      if(b > maxElement){
        maxElement = b;
      }
  }
  //cout << end1-start1 << " -- ILK OKUMA SU (s).\n";

  int nov = maxElement +1;
  //cout <<"nov " << nov << endl;
  //cout <<"nnv " << nnv << endl;
  int * adj  = new int[nnv];
  int * xadj = new int[nov+1];
  xadj[0]=0;

  int j = 0;
  int maxSize = -1;

  for(int i=0; i < nov ; i++ ){
    auto current = hashmap.find(i);
    if (current == hashmap.end()){
        xadj[i+1] = xadj[i];
    }
    else{
        int size = current->second.size();
        maxSize = max(size,maxSize);

        xadj[i+1] = xadj[i] + size;
        for(auto val : current->second) {
            adj[j] = val;
            j++;
        }
    }
  }
  // cout << "maxSize: "<<maxSize<<endl;
  // cout << end-start << " -- OKUMA SURE (s).\n";
  wrapper(xadj,adj,k,nov,nnv);
  //cout<<"CYCLES: --> "<<countCycles_sparse(xadj, adj,k,nov)<<endl;

  /*double end2 = omp_get_wtime();
  cout << end2-start << " -- TOTAL SURE (s).\n";*/
}

int main(int argc, char *argv[]){
    char* fname = argv[1];
    int k = atoi(argv[2]);
    read_mtxbin(fname,k);
    return 0;
}
