//#include "scale.h"

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <math.h> /* fabs */
#include <string.h>
#include <stdlib.h>
#include <sstream>
#include <unordered_map>

using namespace std;
#define THREADS_PER_BLOCK 256

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}


void printArray(int *arr, int nov){
    for (int j=0;j<nov;j++)
    {
        cout << j << " " << arr[j] << endl;
    }
}
__device__ bool check(int marked[], int round, int val){
  for(int i = 0; i < round; i++){
    if(marked[i] == val){return false;}
  }
  return true;
}

__device__ void DFS_sparse(int xadj[], int adj[], int marked[], int n,
         int vert, int start, int &count, int round) //vert: bulundugu konum //start: baslangıc noktası
{
    marked[round] = vert;

    int start_index = xadj[vert];
    int path_length = xadj[vert+1];

    if (n == 0){
      marked[round] = -1;
        for(int i = start_index; i < path_length; i++){
            if(adj[i] == start){
                count++;
                break;
            }
        }
        return;
    }
    for(int i=start_index; i < path_length; i++){
        if(check(marked, round,adj[i])){
            DFS_sparse(xadj, adj, marked, n-1, adj[i], start, count, round + 1);
        }
    }
    marked[round] = -1;
}

__global__ void kernel(int* adj, int* xadj, int* output, int n, int nov){
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  __shared__ int marked[THREADS_PER_BLOCK][10];
  if(index < nov){
      //int *marked = new int[n];
      //memset(marked, -1, n * sizeof(int)); // bu belki silinebilir
      int localcount = 0;
      int round = 0;
      DFS_sparse(xadj, adj, marked[threadIdx.x], n - 1, index, index, localcount, round);
      output[index] = localcount;
    }
}

void wrapper(int *xadj, int *adj, int n,  int nov, int nnz){
  hipSetDevice(0);
  int *adj_d;
  int *xadj_d;
  int *output_d;
  int *output_h = new int[nov];
  int numBlock = (nov + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  hipEvent_t start, stop;
  float elapsedTime;

  gpuErrchk(hipMalloc((void**)&adj_d, (nnz) * sizeof(int)));
  gpuErrchk(hipMalloc((void**)&xadj_d, (nov + 1) * sizeof(int)));

  gpuErrchk(hipMalloc((void**)&output_d, (nov) * sizeof(int)));

  //gpuErrchk(cudaMallocHost((void **)&output_h, (nov) * sizeof(int)));

  gpuErrchk(hipMemcpy(adj_d, adj, (nnz) * sizeof(int), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(xadj_d, xadj, (nov + 1) * sizeof(int), hipMemcpyHostToDevice));

  hipEventCreate(&start);
  hipEventRecord(start, 0);

  kernel<<<numBlock, THREADS_PER_BLOCK>>>(adj_d, xadj_d, output_d, n, nov);

  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipMemcpy(output_h, output_d, (nov) * sizeof(int), hipMemcpyDeviceToHost));
  //printArray(output_h,nov);
  hipEventCreate(&stop);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&elapsedTime, start, stop);
  //printf("GPU scale took: %f s\n", elapsedTime/1000);

  hipFree(adj_d);
  hipFree(xadj_d);
}


void  read_mtxbin(string fname, int k){
  //cout << "fname: " << fname << endl;
  ifstream infile(fname);
  int a, b;
  int counter =0;
  int nnv = 0;
  unordered_map<int, vector<int> > hashmap;

  int  maxElement  = -1;

  while (infile >> a >> b)
  {
      nnv+=2;
      hashmap[a].push_back(b);
      hashmap[b].push_back(a);

      if(b > maxElement){
        maxElement = b;
      }
  }
  //cout << end1-start1 << " -- ILK OKUMA SU (s).\n";

  int nov = maxElement +1;
  //cout <<"nov " << nov << endl;
  //cout <<"nnv " << nnv << endl;
  int * adj  = new int[nnv];
  int * xadj = new int[nov+1];
  xadj[0]=0;

  int j = 0;

  for(int i=0; i < nov ; i++ ){
    auto current = hashmap.find(i);
    if (current == hashmap.end()){
        xadj[i+1] = xadj[i];
    }
    else{
        int size = current->second.size();
        xadj[i+1] = xadj[i] + size;
        for(auto val : current->second) {
            adj[j] = val;
            j++;
        }
    }
  }

  //cout << end-start << " -- OKUMA SURE (s).\n";
  wrapper(xadj,adj,k,nov,nnv);
  //cout<<"CYCLES: --> "<<countCycles_sparse(xadj, adj,k,nov)<<endl;

  /*double end2 = omp_get_wtime();
  cout << end2-start << " -- TOTAL SURE (s).\n";*/
}

int main(int argc, char *argv[]){
    char* fname = argv[1];
    int k = atoi(argv[2]);
    read_mtxbin(fname,k);
    return 0;
}
