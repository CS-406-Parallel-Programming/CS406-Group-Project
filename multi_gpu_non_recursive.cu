//#include "scale.h"

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <math.h> /* fabs */
#include <string.h>
#include <stdlib.h>
#include <sstream>
#include "omp.h"
#include <unordered_map>

using namespace std;
#define THREADS_PER_BLOCK 256
#define PARALEL_THREAD_COUNT 32
#define GPU_MULTIPLIER 10
#define PARALEL_CPU 28

// #define VIRTUAL_THREAD_COUNT 68
// 4  10 40
// 28 1  28

int flag;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}

void DFS_sparse(int xadj[], int adj[], bool marked[], int n,
         int vert, int start, int &count) //vert: bulundugu konum //start: baslangıc noktası
{
    marked[vert] = true;
    int start_index = xadj[vert];
    int path_length = xadj[vert+1];

    if (n == 0){
        marked[vert] = false;
        for(int i = start_index; i < path_length; i++){
            if(adj[i] == start){
                count++;
                break;
            }
        }
        return;
    }

// if(path_length-start_index <=1) return;

    for(int i=start_index; i < path_length; i++){
        if(!marked[adj[i]]){
            DFS_sparse(xadj, adj, marked, n-1, adj[i], start, count);
        }
    }
    marked[vert] = false;
}

void printArray(int *arr, int nov){
    for (int j=0;j<nov;j++)
    {
        cout << j << " " << arr[j] << endl;
    }
}
              
__global__ void kernel3(int* adj, int* xadj, int* output, int nov, int novStart){

  int index = novStart + threadIdx.x + (blockIdx.x * blockDim.x);
  if(index < nov){
      // if(index ==0)printf("called gpu \n");
      //int *marked = new int[n];
      //memset(marked, -1, n * sizeof(int)); // bu belki silinebilir
      int localcount = 0;
      // int round = 0;

      // 0-->
      int s0 = xadj[index];
      int e0 = xadj[index+1];

      for(int i=s0; i < e0; i++){
        // 0 --> 1

        int neighbour_1 = adj[i];
        int s1  = xadj[neighbour_1];
        int e1  = xadj[neighbour_1+1];

        for(int j=s1;j < e1; j++){
          // 0 --> 1 --> 2

          int neighbour_2 =  adj[j];
          if (neighbour_2 == index) continue;
          int s2  = xadj[neighbour_2];
          int e2  = xadj[neighbour_2+1];

          for(int k=s2; k < e2; k++){

            // 0 --> 1 --> 2 --> 3
            int neighbour_3 =  adj[k];
            if (neighbour_3 == index){
              localcount+=1;
              break;
            }
          }
        }
      }
      output[index-novStart] = localcount;
    }
}

__global__ void kernel4(int* adj, int* xadj, int* output, int nov,int novStart ){
  int index =novStart + threadIdx.x + (blockIdx.x * blockDim.x);
  if(index < nov){
      int localcount = 0;

      int s0 = xadj[index];
      int e0 = xadj[index+1];

      for(int i=s0; i < e0; i++){

        int neighbour_1 = adj[i];
        int s1  = xadj[neighbour_1];
        int e1  = xadj[neighbour_1+1];

        for(int j=s1;j < e1; j++){

          int neighbour_2 =  adj[j];
          if (neighbour_2 == index) continue;
          int s2  = xadj[neighbour_2];
          int e2  = xadj[neighbour_2+1];

          for(int k=s2; k < e2; k++){

            // 0 --> 1 --> 2 --> 3

            int neighbour_3 =  adj[k];
            //eliminate 3 == 0
            if (neighbour_3 == index) continue;
            // eliminate 3 ==1
            if (neighbour_3 == neighbour_1) continue;
            int s3  = xadj[neighbour_3];
            int e3  = xadj[neighbour_3+1];


            for(int n=s3; n < e3; n++){
              //0 -->1 -->2 -->3 -->4


              int neighbour_4 =  adj[n];
              if (neighbour_4 == index){
                localcount+=1;
                break;
              }
            }
          }
        }
      }
      output[index-novStart] = localcount;
    }
}
__global__ void kernel5(int* adj, int* xadj, int* output, int nov, int novStart){
  int index = novStart + threadIdx.x + (blockIdx.x * blockDim.x);
  if(index < nov){
      //int *marked = new int[n];
      //memset(marked, -1, n * sizeof(int)); // bu belki silinebilir
      int localcount = 0;
      // int round = 0;

      // 0-->
      int s0 = xadj[index];
      int e0 = xadj[index+1];

      for(int i=s0; i < e0; i++){
        // 0 --> 1

        int neighbour_1 = adj[i];
        int s1  = xadj[neighbour_1];
        int e1  = xadj[neighbour_1+1];

        for(int j=s1;j < e1; j++){
          // 0 --> 1 --> 2

          int neighbour_2 =  adj[j];
          //eliminate 0 == 2
          if (neighbour_2 == index) continue;
          int s2  = xadj[neighbour_2];
          int e2  = xadj[neighbour_2+1];

          for(int k=s2; k < e2; k++){

            // 0 --> 1 --> 2 --> 3

            int neighbour_3 =  adj[k];
            //eliminate 3 == 0
            if (neighbour_3 == index) continue;
            // eliminate 3 ==1
            if (neighbour_3 == neighbour_1) continue;
            int s3  = xadj[neighbour_3];
            int e3  = xadj[neighbour_3+1];


            for(int n=s3; n < e3; n++){
              //0 -->1 -->2 -->3 -->4


              int neighbour_4 =  adj[n];
              //eliminate 4 == 0
              if (neighbour_4 == index) continue;
              // eliminate 4 ==1
              if (neighbour_4 == neighbour_1) continue;
              // eliminate 4 ==2
              if (neighbour_4 == neighbour_2) continue;

              int s4  = xadj[neighbour_4];
              int e4  = xadj[neighbour_4+1];


              for(int o=s4; o < e4; o++){

                //0 -->1 -->2 -->3 -->4--> 5

                int neighbour_5 =  adj[o];
                if (neighbour_5 == index){
                  localcount+=1;
                  break;
                }
              }
            }
          }
        }
      }
      output[index-novStart] = localcount;
    }
}


void wrapper(int *xadj, int *adj, int n,  int nov, int nnz){

  int *output_h = new int[nov];

 // double start_cpu, end_cpu;
 // start_cpu = omp_get_wtime();


  #pragma omp parallel num_threads(PARALEL_THREAD_COUNT)
  {

    int threadId=omp_get_thread_num ();
    // cout<< threadId<<endl;

    int virtual_thread_count  = GPU_MULTIPLIER *4 + PARALEL_CPU;
    int novForThread = (nov+virtual_thread_count-1)/virtual_thread_count;



    if(threadId <=3)
    {
      int novStart = GPU_MULTIPLIER * novForThread * threadId;
      int novEnd   = GPU_MULTIPLIER * novForThread * (threadId+1);
      if (novEnd > nov) novEnd = nov;
      int numBlock = (novEnd-novStart + THREADS_PER_BLOCK-1) / THREADS_PER_BLOCK;
      // printf("nov s %d e %d \n", novStart,novEnd);


      hipSetDevice(threadId);

      int *adj_d;
      int *xadj_d;
      int *output_d;
      hipEvent_t start, stop;
      float elapsedTime;

      gpuErrchk(hipMalloc((void**)&adj_d, (nnz) * sizeof(int)));
      gpuErrchk(hipMalloc((void**)&xadj_d, (nov + 1) * sizeof(int)));

      gpuErrchk(hipMalloc((void**)&output_d, (novEnd-novStart) * sizeof(int)));

      //gpuErrchk(cudaMallocHost((void **)&output_h, (nov) * sizeof(int)));

      gpuErrchk(hipMemcpy(adj_d, adj, (nnz) * sizeof(int), hipMemcpyHostToDevice));
      gpuErrchk(hipMemcpy(xadj_d, xadj, (nov + 1) * sizeof(int), hipMemcpyHostToDevice));

      hipEventCreate(&start);
      hipEventRecord(start, 0);
      double start_gpu = omp_get_wtime();
      hipStream_t stream1;
      hipStreamCreate ( &stream1) ;

        // printf("threadId entry to kernel %d GPU \n", threadId );
      if      (n==3)kernel3<<<numBlock, THREADS_PER_BLOCK,0,stream1>>>(adj_d, xadj_d, output_d, novEnd,novStart);
      else if (n==4)kernel4<<<numBlock, THREADS_PER_BLOCK,0,stream1>>>(adj_d, xadj_d, output_d, novEnd,novStart);
       else if (n==5)kernel5<<<numBlock, THREADS_PER_BLOCK,0,stream1>>>(adj_d, xadj_d, output_d, novEnd,novStart);
      //combination<<<numBlocks, threadsPerBlock>>>(adj_d, xadj_d, output_d, n, nov);
      // printf("threadId exit to kernel %d GPU \n", threadId );
   //   double end_gpu = omp_get_wtime();


      gpuErrchk(hipDeviceSynchronize());
      hipEventCreate(&stop);
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&elapsedTime, start, stop);

      if(flag == 1)	printf("GPU scale took: %f s on gpu  %d \n", elapsedTime/1000, threadId);

      gpuErrchk(hipMemcpy(output_h+novStart, output_d, (novEnd-novStart) * sizeof(int), hipMemcpyDeviceToHost));
      hipFree(adj_d);
      hipFree(xadj_d);


    }
    else{
      // printf("Entered \n" );

      // int novForThread = (nov+PARALEL_THREAD_COUNT-1)/PARALEL_THREAD_COUNT;
      // int novStart = novForThread * threadId;
      // int novEnd   = novForThread * (threadId+1);
      // if (novEnd> nov) novEnd = nov;
      // int numBlock = (novEnd-novStart + THREADS_PER_BLOCK-1) / THREADS_PER_BLOCK;
      int novStart = 4 * GPU_MULTIPLIER*novForThread +  1 * novForThread * (threadId-4);
      int novEnd   =  novStart + 1*  novForThread ;
      if (novEnd> nov) novEnd = nov;


      // printf("nov s %d e %d  -cpu \n", novStart,novEnd);

      bool *marked = new bool[nov];
      memset(marked, false, nov * sizeof(bool)); // bu belki silinebilir

      double start_thread = omp_get_wtime();
      for(int i = novStart; i < novEnd; i++){
          int localcount = 0;
          DFS_sparse(xadj, adj, marked, n - 1, i, i, localcount);
          output_h[i ] = localcount;


      }
      double end_thread = omp_get_wtime();
      printf("Took %f secs \n", end_thread -start_thread );


    }


  }
//  end_cpu = omp_get_wtime();

   // printf("Took %f secs \n", end_cpu - start_cpu);
  	if(flag == 0)	printArray(output_h,nov);
}


void  read_mtxbin(string fname, int k){
  //cout << "fname: " << fname << endl;
  ifstream infile(fname);
  int a, b;
  int nnv = 0;
  unordered_map<int, vector<int> > hashmap;

  int  maxElement  = -1;

  while (infile >> a >> b)
  {
      nnv+=2;
      hashmap[a].push_back(b);
      hashmap[b].push_back(a);

      if(b > maxElement){
        maxElement = b;
      }
  }
  //cout << end1-start1 << " -- ILK OKUMA SU (s).\n";

  int nov = maxElement +1;
  //cout <<"nov " << nov << endl;
  //cout <<"nnv " << nnv << endl;
  int * adj  = new int[nnv];
  int * xadj = new int[nov+1];
  xadj[0]=0;

  int j = 0;
  int maxSize = -1;

  for(int i=0; i < nov ; i++ ){
    auto current = hashmap.find(i);
    if (current == hashmap.end()){
        xadj[i+1] = xadj[i];
    }
    else{
        int size = current->second.size();
        maxSize = max(size,maxSize);

        xadj[i+1] = xadj[i] + size;
        for(auto val : current->second) {
            adj[j] = val;
            j++;
        }
    }
  }
  // cout << "maxSize: "<<maxSize<<endl;
  // cout << end-start << " -- OKUMA SURE (s).\n";
  wrapper(xadj,adj,k,nov,nnv);
  //cout<<"CYCLES: --> "<<countCycles_sparse(xadj, adj,k,nov)<<endl;

  /*double end2 = omp_get_wtime();
  cout << end2-start << " -- TOTAL SURE (s).\n";*/
}

int main(int argc, char *argv[]){
    char* fname = argv[1];
    int k = atoi(argv[2]);
    flag = atoi(argv[3]);
    read_mtxbin(fname,k);
    return 0;
}
