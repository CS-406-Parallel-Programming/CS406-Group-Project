#include "hip/hip_runtime.h"
//#include "scale.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <math.h> /* fabs */
#include <string.h>
#include <stdlib.h>
#include <sstream>
#include <unordered_map>

using namespace std;
#define THREADS_PER_BLOCK 256

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}


void printArray(int *arr, int nov){
    for (int j=0;j<nov;j++)
    {
        cout << j << " " << arr[j] << endl;
    }
}
__device__ bool check(int marked[], int round, int val){
  for(int i = 0; i < round; i++){
    if(marked[i] == val){return false;}
  }
  return true;
}

__device__ void DFS_sparse(int xadj[], int adj[], int marked[], int n,
         int vert, int start, int &count, int round) //vert: bulundugu konum //start: baslangıc noktası
{
    marked[round] = vert;

    int start_index = xadj[vert];
    int path_length = xadj[vert+1];

    if (n == 0){
      marked[round] = -1;
        for(int i = start_index; i < path_length; i++){
            if(adj[i] == start){
                count++;
                break;
            }
        }
        return;
    }
    for(int i=start_index; i < path_length; i++){
        if(check(marked, round,adj[i])){
            DFS_sparse(xadj, adj, marked, n-1, adj[i], start, count, round + 1);
        }
    }
    marked[round] = -1;
}

__global__ void kernel(int* adj, int* xadj, int* output, int n, int nov){
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  __shared__ int marked[THREADS_PER_BLOCK][10];
  if(index < nov){
      //int *marked = new int[n];
      //memset(marked, -1, n * sizeof(int)); // bu belki silinebilir
      int localcount = 0;
      int round = 0;
      DFS_sparse(xadj, adj, marked[threadIdx.x], n - 1, index, index, localcount, round);
      output[index] = localcount;
    }
}

__global__ void combination(int* adj, int* xadj, int* output, int n, int nov){


  int v =  blockDim.x * blockIdx.x +  threadIdx.x;
  int i =  blockDim.y * blockIdx.y +  threadIdx.y;
  int j =  blockDim.z * blockIdx.z +  threadIdx.z;

 // accessVertex
  // if (v==0 && i == 0 && j ==0) {printf("running");}
  int start = adj[v];
  int end   = adj[v+1];

  int neighbourCount = end-start;


  if (i>=neighbourCount) return;
  int first_neighbour = xadj[adj[start + i]];

  int first_neighbour_start = adj[first_neighbour];
  int first_neighbour_end = adj[first_neighbour+1];
  int first_neighbour_neighbourCount = first_neighbour_end-first_neighbour_start;

  if(j>=first_neighbour_neighbourCount) return;

  int second_neighbour = xadj[adj[first_neighbour_start+j]];


  // check all of them unique
  if (v == second_neighbour) return;

  // check first -> second -> third has connection


  for(int k = start; k < end; k ++){
     if (xadj[adj[start + k]] == second_neighbour){
       AtomicAdd(&output[v],1);
       break;
     }
  }
}


void wrapper(int *xadj, int *adj, int n,  int nov, int nnz,int maxSize){

  int X = nov;
  int Y = maxSize;
  int Z = maxSize;

  dim3 threadsPerBlock(8, 8, 8);
  dim3 numBlocks(X/threadsPerBlock.x,  /* for instance 512/8 = 64*/
                 Y /threadsPerBlock.y,
                 Z/threadsPerBlock.z);







  hipSetDevice(0);
  int *adj_d;
  int *xadj_d;
  int *output_d;
  int *output_h = new int[nov];
  int numBlock = (nov + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  hipEvent_t start, stop;
  float elapsedTime;

  gpuErrchk(hipMalloc((void**)&adj_d, (nnz) * sizeof(int)));
  gpuErrchk(hipMalloc((void**)&xadj_d, (nov + 1) * sizeof(int)));

  gpuErrchk(hipMalloc((void**)&output_d, (nov) * sizeof(int)));

  //gpuErrchk(hipHostMalloc((void **)&output_h, (nov) * sizeof(int)));

  gpuErrchk(hipMemcpy(adj_d, adj, (nnz) * sizeof(int), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(xadj_d, xadj, (nov + 1) * sizeof(int), hipMemcpyHostToDevice));

  hipEventCreate(&start);
  hipEventRecord(start, 0);

  // kernel<<<numBlock, THREADS_PER_BLOCK>>>(adj_d, xadj_d, output_d, n, nov);
  combination<<<numBlocks, threadsPerBlock>>>(adj_d, xadj_d, output_d, n, nov);

  gpuErrchk(hipDeviceSynchronize());

  gpuErrchk(hipMemcpy(output_h, output_d, (nov) * sizeof(int), hipMemcpyDeviceToHost));
  printArray(output_h,nov);
  hipEventCreate(&stop);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("GPU scale took: %f s\n", elapsedTime/1000);

  hipFree(adj_d);
  hipFree(xadj_d);
}


void  read_mtxbin(string fname, int k){
  //cout << "fname: " << fname << endl;
  ifstream infile(fname);
  int a, b;
  int counter =0;
  int nnv = 0;
  unordered_map<int, vector<int> > hashmap;

  int  maxElement  = -1;

  while (infile >> a >> b)
  {
      nnv+=2;
      hashmap[a].push_back(b);
      hashmap[b].push_back(a);

      if(b > maxElement){
        maxElement = b;
      }
  }
  //cout << end1-start1 << " -- ILK OKUMA SU (s).\n";

  int nov = maxElement +1;
  //cout <<"nov " << nov << endl;
  //cout <<"nnv " << nnv << endl;
  int * adj  = new int[nnv];
  int * xadj = new int[nov+1];
  xadj[0]=0;

  int j = 0;
  int maxSize = -1;

  for(int i=0; i < nov ; i++ ){
    auto current = hashmap.find(i);
    if (current == hashmap.end()){
        xadj[i+1] = xadj[i];
    }
    else{
        int size = current->second.size();
        maxSize = max(size,maxSize);

        xadj[i+1] = xadj[i] + size;
        for(auto val : current->second) {
            adj[j] = val;
            j++;
        }
    }
  }
  cout << "maxSize: "<<maxSize<<endl;
  // cout << end-start << " -- OKUMA SURE (s).\n";
  wrapper(xadj,adj,k,nov,nnv,maxSize);
  //cout<<"CYCLES: --> "<<countCycles_sparse(xadj, adj,k,nov)<<endl;

  /*double end2 = omp_get_wtime();
  cout << end2-start << " -- TOTAL SURE (s).\n";*/
}

int main(int argc, char *argv[]){
    char* fname = argv[1];
    int k = atoi(argv[2]);
    read_mtxbin(fname,k);
    return 0;
}
